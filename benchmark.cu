#include <fstream>
#include <vector>
#include <iostream>
#include <string>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <chrono>

void load_matrix(const std::string& filename, std::vector<double>& data, const int64_t instance_size) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Cannot open file\n";
        throw std::runtime_error("Cannot open file");
    }

    data.resize(instance_size * instance_size);
    file.read(reinterpret_cast<char*>(data.data()), instance_size * instance_size * sizeof(double));
}

// Check if the function returns a CUDA error
#define CHECK_CUDA(func)                                                       \
do {                                                                           \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at %s:%d with error: %s (%d)",                 \
               __FILE__, __LINE__, hipGetErrorString(status), status);        \
        std::cout << std::endl;                                                \
    }                                                                          \
} while (0) // wrap it in a do-while loop to be called with a semicolon

// Check if the function returns a cuBLAS error
#define CHECK_CUBLAS(func)                                                     \
do {                                                                           \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("cuBLAS error %d at %s:%d", status, __FILE__, __LINE__);        \
        std::cout << std::endl;                                                \
    }                                                                          \
} while (0)

// Check if the function returns a cuSPARSE error
#define CHECK_CUSOLVER(func)                                                   \
do {                                                                           \
    hipsolverStatus_t status = (func);                                          \
    if (status != HIPSOLVER_STATUS_SUCCESS) {                                   \
        printf("cuSOLVER error %d at %s:%d", status, __FILE__, __LINE__);      \
        std::cout << std::endl;                                                \
    }                                                                          \
} while (0)

// Check if the function returns a cuSPARSE error
#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("cuSPARSE error %s (%d) at %s:%d",                              \
              hipsparseGetErrorString(status), status, __FILE__, __LINE__);     \
        std::cout << std::endl;                                                \
    }                                                                          \
}

std::chrono::duration<double> cusolver_FP64_psd( hipsolverHandle_t solverH, hipblasHandle_t cublasH, double* dA, double* dA_psd, size_t n) {
    int *devInfo; CHECK_CUDA(hipMalloc(&devInfo, sizeof(int)));
    size_t nn = n * n;
    double one_d = 1.0;
    double zero_d = 0.0;

    auto start = std::chrono::high_resolution_clock::now();
    double *dW; CHECK_CUDA(hipMalloc(&dW, n*sizeof(double)));
    int lwork_ev = 0;
    CHECK_CUSOLVER(hipsolverDnDsyevd_bufferSize(
        solverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
        n, dA, n, dW, &lwork_ev));
    double *dWork_ev; CHECK_CUDA(hipMalloc(&dWork_ev, lwork_ev*sizeof(double)));
    CHECK_CUSOLVER(hipsolverDnDsyevd(
        solverH,
        HIPSOLVER_EIG_MODE_VECTOR,
        HIPBLAS_FILL_MODE_UPPER,
        n, dA, n, dW,
        dWork_ev, lwork_ev, devInfo));
    CHECK_CUDA(hipDeviceSynchronize());

    std::vector<double> W_h(n);
    CHECK_CUDA(hipMemcpy(W_h.data(), dW, n*sizeof(double), hipMemcpyDeviceToHost));
    for(int i=0;i<n;i++) if(W_h[i]<0) W_h[i]=0;

    // Copy eigenvectors from dA to dV
    double *dV; CHECK_CUDA(hipMalloc(&dV, nn*sizeof(double)));
    CHECK_CUDA(hipMemcpy(dV, dA, nn*sizeof(double), hipMemcpyDeviceToDevice));

    // Scale columns of dV by W_h
    for(int i=0;i<n;i++){
        CHECK_CUBLAS(hipblasDscal(cublasH, n, &W_h[i], dV + i*n, 1));
    }

    // Reconstruct A_psd = V * V^T
    double *dTmp; CHECK_CUDA(hipMalloc(&dTmp, nn*sizeof(double)));
    CHECK_CUBLAS(hipblasGemmEx(
        cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n, n, n,
        &one_d,
        dV, HIP_R_64F, n,
        dA, HIP_R_64F, n,
        &zero_d,
        dTmp, HIP_R_64F, n,
        HIP_R_64F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    CHECK_CUDA(hipMemcpy(dA_psd, dTmp, nn*sizeof(double), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipFree(dTmp));
    CHECK_CUDA(hipFree(dV));
    auto end = std::chrono::high_resolution_clock::now();
    return end - start;
}

int main(int argc, char* argv[]) {
    std::vector<std::string> datasets;
    std::vector<size_t> instance_sizes;
    int restarts = 1;

    // Parse command line arguments
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if (arg == "--datasets") {
            ++i;
            while (i < argc && argv[i][0] != '-') {
                datasets.push_back(argv[i]);
                ++i;
            }
            --i;
        } else if (arg == "--instance_sizes") {
            ++i;
            while (i < argc && argv[i][0] != '-') {
                instance_sizes.push_back(std::stoul(argv[i]));
                ++i;
            }
            --i;
        } else if (arg == "--restarts") {
            if (i + 1 < argc) {
                restarts = std::stoi(argv[++i]);
            }
        }
    }

    /* Initialize data and handles */
    std::vector<double> data;
    hipsolverHandle_t solverH;
    CHECK_CUSOLVER(hipsolverDnCreate(&solverH));
    hipblasHandle_t cublasH;
    CHECK_CUBLAS(hipblasCreate(&cublasH));

    /* Warmup the GPU */
    std::cout << "Warming up the GPU...";
    for (int i = 0; i < restarts; ++i) {
        double *dA, *dB, *dC;
        size_t n = 1024; // Use a fixed size for warmup
        double one_d = 1.0;
        double zero_d = 0.0;
        CHECK_CUDA(hipMalloc(&dA, n * n * sizeof(double)));
        CHECK_CUDA(hipMalloc(&dB, n * n * sizeof(double)));
        CHECK_CUDA(hipMalloc(&dC, n * n * sizeof(double)));
        // Initialize dA and dB with ones
        CHECK_CUDA(hipMemset(dA, 1, n * n * sizeof(double)));
        CHECK_CUDA(hipMemset(dB, 1, n * n * sizeof(double)));
        CHECK_CUBLAS(hipblasDgemm(
            cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, n, n,
            &one_d, dA, n, dB, n,
            &zero_d, dC, n));
        CHECK_CUDA(hipFree(dA));
        CHECK_CUDA(hipFree(dB));
        CHECK_CUDA(hipFree(dC));
    }
    std::cout << " done." << std::endl << std::endl;

    /* Main benchmarking loop */
    for (const auto& dataset : datasets) {
        for (const auto& n : instance_sizes) {
            /* 0) Generate the matrix */
            std::cout << "DATASET '" << dataset << "' WITH INSTANCE SIZE " << n << std::endl;

            // load the matrix from the generated binary file
            std::string filename = "data/bin/" + dataset + "-" + std::to_string(n) + ".bin";
            load_matrix(filename, data, n);

            // copy the matrix to the device
            double *A, *A_psd, *A_psd_ref;
            CHECK_CUDA(hipMalloc(&A,         n * n * sizeof(double)));
            CHECK_CUDA(hipMalloc(&A_psd,     n * n * sizeof(double)));
            CHECK_CUDA(hipMalloc(&A_psd_ref, n * n * sizeof(double)));
            CHECK_CUDA(hipMemcpy(A, data.data(), n * n * sizeof(double), hipMemcpyHostToDevice));

            /* 1) Pure GEMM and EIG times */
            std::cout << "\t Pure EIG and GEMM times" << std::endl;
            // cuSOLVER FP32
            // cuSOLVER FP64 eig
            // TF16
            // TF32
            // FP32
            // FP64

            /* 2) PSD cone projection */
            std::cout << "\t PSD cone projection" << std::endl;

            // cuSOLVER FP64
            std::chrono::duration<double> duration(0.0);
            for (int i = 0; i < restarts; ++i) {
                duration += cusolver_FP64_psd(solverH, cublasH, A, A_psd_ref, n);
            }
            duration /= restarts;
            std::cout << "\t\t cuSOLVER FP64 -- Time: " << std::scientific << duration.count() << " s" << std::endl;
            std::cout << "\t\t        Relative error: 0.0" << std::endl;

            // cuSOLVER FP32
            // composite TF16
            // composite TF32
            // composite FP32
            // composite FP64
        }
    }

    /* Clean up */
    CHECK_CUSOLVER(hipsolverDnDestroy(solverH));

    return 0;
}